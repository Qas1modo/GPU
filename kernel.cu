#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 32
#define FULL_MASK 0xFFFFFFFF
#define TILE_SIZE_HALF 16


__global__ void calculateAccount(int *changes, int* account, int* sum, int clients, int periods) {
    __shared__ int prevBlockData[TILE_SIZE][TILE_SIZE + 1];
    __shared__ int blockData[TILE_SIZE][TILE_SIZE + 1];
    __shared__ int nextBlockData[TILE_SIZE][TILE_SIZE + 1];
    int* prevBlock = &prevBlockData[0][0];
    int* block = &blockData[0][0];
    int* nextBlock = &nextBlockData[0][0];
    int threadSum = 0;
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int tile_index = (threadIdx.y % TILE_SIZE_HALF) * (TILE_SIZE + 1) + threadIdx.x;
    int in_index = threadIdx.y * clients + x;
    int index_out = (threadIdx.y % TILE_SIZE_HALF) * clients + x;
    prevBlockData[threadIdx.y][threadIdx.x] = changes[in_index];
    in_index += TILE_SIZE * clients;
    __syncthreads();
    if (threadIdx.y == TILE_SIZE_HALF) {       
        for (int threadRow = 0; threadRow < TILE_SIZE; threadRow++) {
            threadSum += prevBlock[threadRow * (TILE_SIZE + 1) + threadIdx.x];
            prevBlock[threadRow * (TILE_SIZE + 1) + threadIdx.x] = threadSum;
        }
    }
    blockData[threadIdx.y][threadIdx.x] = changes[in_index];
    in_index += TILE_SIZE * clients;
    __syncthreads();
    for (int block_Y = 2 * TILE_SIZE; block_Y < periods; block_Y+=TILE_SIZE) {
        if (threadIdx.y < TILE_SIZE_HALF) {
            nextBlock[tile_index] = changes[in_index];
            in_index += TILE_SIZE_HALF * clients;
            nextBlock[tile_index + (TILE_SIZE_HALF) * (TILE_SIZE + 1)] = changes[in_index];
            in_index += TILE_SIZE_HALF * clients;
        } else {
            if (threadIdx.y == TILE_SIZE_HALF) {
                for (int threadRow = 0; threadRow < TILE_SIZE * (TILE_SIZE + 1); threadRow += TILE_SIZE + 1) {
                    threadSum += block[threadRow + threadIdx.x];
                    block[threadRow + threadIdx.x] = threadSum;
                }
            }
            int threadRowSum = prevBlock[tile_index];
            int y = (index_out - x) / clients;
            account[index_out] = threadRowSum;
            for (int offset = warpSize/2; offset > 0; offset /= 2) threadRowSum += __shfl_down_sync(FULL_MASK, threadRowSum, offset);
            if (threadIdx.x == 0) {
                atomicAdd(&sum[y], threadRowSum);
            }
            index_out += TILE_SIZE_HALF * clients; 
            threadRowSum = prevBlock[tile_index + TILE_SIZE_HALF * (TILE_SIZE + 1)];
            account[index_out] = threadRowSum;
            for (int offset = warpSize/2; offset > 0; offset /= 2) threadRowSum += __shfl_down_sync(FULL_MASK, threadRowSum, offset);
            if (threadIdx.x == 0) {
                atomicAdd(&sum[y + TILE_SIZE_HALF], threadRowSum);
            }
            index_out += TILE_SIZE_HALF * clients; 
        }
        __syncthreads();
        int* temp = prevBlock;
        prevBlock = block;
        block = nextBlock;
        nextBlock = temp;
    }
    if (threadIdx.y == TILE_SIZE_HALF) {
        for (int threadRow = 0; threadRow < TILE_SIZE * (TILE_SIZE + 1); threadRow += TILE_SIZE + 1) {
            threadSum += block[threadRow + threadIdx.x];
            block[threadRow + threadIdx.x] = threadSum;
        }
    }
    tile_index = threadIdx.y * (TILE_SIZE + 1) + threadIdx.x;
    threadSum = prevBlock[tile_index];
    account[(threadIdx.y + periods - 2 * TILE_SIZE) * clients + x] = threadSum;
    for (int offset = warpSize/2; offset > 0; offset /= 2) threadSum += __shfl_down_sync(FULL_MASK, threadSum, offset);
    if (threadIdx.x == 0) {
        atomicAdd(&sum[periods - 2 *TILE_SIZE + threadIdx.y], threadSum);
    }
    __syncthreads();
    threadSum = block[tile_index];
    account[(threadIdx.y + periods - TILE_SIZE) * clients + x] = threadSum;
    for (int offset = warpSize/2; offset > 0; offset /= 2) threadSum += __shfl_down_sync(FULL_MASK, threadSum, offset);
    if (threadIdx.x == 0) {
        atomicAdd(&sum[periods - TILE_SIZE + threadIdx.y], threadSum);
    }
}


void solveGPU(int *changes, int *account, int *sum, int clients, int periods) {
    dim3 numThreads(TILE_SIZE, TILE_SIZE);
    calculateAccount<<<clients / TILE_SIZE, numThreads>>>(changes, account, sum, clients, periods);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }
}

